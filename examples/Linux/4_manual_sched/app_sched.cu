#include "hip/hip_runtime.h"
#include <chrono>
#include <thread>
#include <random>
#include <thread>
#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>

#include "xsched/xsched.h"
#include "xsched/cuda/hal.h"

#define VECTOR_SIZE (1 << 25) // 32MB
#define N 100    // Number of vector additions per task
#define M 10000  // Number of tasks, (almost) never stops

// 1 is for low priority, 2 is for high priority
hipStream_t stream_1, stream_2;
HwQueueHandle hwq_1, hwq_2;
XQueueHandle xq_1, xq_2;

__global__ void vector_add(const float* A, const float* B, float* C, int n)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= n) return;
    C[i] = A[i] + B[i];
}

void task(float *d_A, float *d_B, float *d_C, hipStream_t stream)
{
    // Launch kernel N times
    int block_size = 256;
    int grid_size = (VECTOR_SIZE + block_size - 1) / block_size;
    
    for (int i = 0; i < N; ++i) {
        vector_add<<<grid_size, block_size, 0, stream>>>(d_A, d_B, d_C, VECTOR_SIZE);
    }
    hipStreamSynchronize(stream);
}

void run(bool is_high_priority)
{
    hipStream_t stream = is_high_priority ? stream_2 : stream_1;

    // Global memory pointers
    float *h_A, *h_B, *h_C;
    float *d_A, *d_B, *d_C;

    size_t size = VECTOR_SIZE * sizeof(float);

    // Allocate host memory
    h_A = (float*)malloc(size);
    h_B = (float*)malloc(size);
    h_C = (float*)malloc(size);

    // Initialize host vectors
    for (int i = 0; i < VECTOR_SIZE; ++i) {
        h_A[i] = static_cast<float>(rand()) / RAND_MAX;
        h_B[i] = static_cast<float>(rand()) / RAND_MAX;
    }

    // Allocate device memory
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // Copy vectors to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Run tasks
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_int_distribution<> dis(30, 50);

    for (int i = 0; i < M; ++i) {
        auto start = std::chrono::high_resolution_clock::now();
        // If this is the high-priority task,
        // suspend the low-priority task when the high-priority task starts.
        if (is_high_priority) XQueueSuspend(xq_1, 0);
        task(d_A, d_B, d_C, stream);
        // If this is the high-priority task,
        // resume the low-priority task when the high-priority task finishes.
        if (is_high_priority) XQueueResume(xq_1, 0);
        auto end = std::chrono::high_resolution_clock::now();
        auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
        printf("%s prio Task %d completed in %ld ms\n", is_high_priority ? "high" : "low ", i, duration.count());

        // Sleep for random interval between tasks
        std::this_thread::sleep_for(std::chrono::milliseconds(dis(gen)));
    }

    // Free memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);
}

int main()
{
    hipStreamCreate(&stream_1);
    CudaQueueCreate(&hwq_1, stream_1);
    XQueueCreate(&xq_1, hwq_1, kPreemptLevelBlock, kQueueCreateFlagNone);
    XQueueSetLaunchConfig(xq_1, 8, 4);

    hipStreamCreate(&stream_2);
    CudaQueueCreate(&hwq_2, stream_2);
    XQueueCreate(&xq_2, hwq_2, kPreemptLevelBlock, kQueueCreateFlagNone);
    XQueueSetLaunchConfig(xq_2, 8, 4);

    // run two tasks within one process
    std::thread thread_lp(run, false);
    std::this_thread::sleep_for(std::chrono::seconds(1));
    std::thread thread_hp(run, true);

    thread_lp.join();
    thread_hp.join();

    return 0;
}
